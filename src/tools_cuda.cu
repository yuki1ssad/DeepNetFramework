#include "tools_cuda.h"


std::map<
    std::string,
    GPU_TICKTOCK
> GPU_TICKTOCKS;

void check_device_data(float* p_data, size_t ele) {
  VLOG(8) << "check_device_data";
  float t[ele];
  checkCudaErrors(hipMemcpy(t, p_data, ele * sizeof(float), hipMemcpyDeviceToHost));
  for (int i = 0; i < ele; i++) {
    VLOG(8) << t[i];
  }
}

std::ostream& operator<<(std::ostream& os, const dim3& dm) {
    os << "dim3(" << dm.x << ", " << dm.y << ", " << dm.z << ")";
    return os;
}



void GPU_TICK(std::string task, hipStream_t stm){
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    GPU_TICKTOCKS[task] = GPU_TICKTOCK();
    GPU_TICKTOCKS[task].tick = start;
    GPU_TICKTOCKS[task].tock = stop;
    checkCudaErrors(hipEventRecord(start, stm));
}

void GPU_TOCK(std::string task, hipStream_t stm){
    checkCudaErrors(hipEventRecord(GPU_TICKTOCKS[task].tock, stm));
    checkCudaErrors(hipEventSynchronize(GPU_TICKTOCKS[task].tock));
    checkCudaErrors(
        hipEventElapsedTime(
            &GPU_TICKTOCKS[task].interval,
            GPU_TICKTOCKS[task].tick,
            GPU_TICKTOCKS[task].tock
        )
    );
}