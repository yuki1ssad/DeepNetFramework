#include "hip/hip_runtime.h"
#include <random>
#include <hipblas.h>
#include <glog/logging.h>
#include <gtest/gtest.h>

#include "Tensor.h"
#include "tools_cuda.h"
#include "kernel_matmul.h"
#include "tools_common.h"

class test_matmul:
    public testing::TestWithParam<
        std::tuple<
            int,  // m
            int,  // n
            int,  // k
            std::function<float(const std::vector<int>&)>,  // W gen
            std::function<float(const std::vector<int>&)>,  // X gen
            dim3  // block
        >
    >
{
public:
    int m ,n, k;
    std::function<float(const std::vector<int>&)> W_gen, X_gen;
    dim3 BLOCK;
    
    float alpha = 1.f, beta = 0.f;
    size_t W_size, X_size, Y_size;
    float *W_host, *X_host, *Y_ground_truth_host, *Y_predict_host, *W_device, *X_device, *Y_ground_truth_device, *Y_predict_device;
    dim3 GRID;
    size_t shared_mem;

    hipblasHandle_t handle = nullptr;

    test_matmul();
    ~test_matmul();
};

test_matmul::test_matmul() {
    std::tie(
        m,
        n,
        k,
        W_gen,
        X_gen,
        BLOCK
    ) = GetParam();

    hipblasCreate(&handle);
    W_size = m * k * sizeof(float);
    X_size = k * n * sizeof(float);
    Y_size = m * n * sizeof(float);

    W_host = (float*)malloc(W_size);
    X_host = (float*)malloc(X_size);
    Y_ground_truth_host = (float*)malloc(Y_size);
    Y_predict_host = (float*)malloc(Y_size);
    checkCudaErrors(hipMalloc(&W_device, W_size));
    checkCudaErrors(hipMalloc(&X_device, X_size));
    checkCudaErrors(hipMalloc(&Y_ground_truth_device, Y_size));
    checkCudaErrors(hipMalloc(&Y_predict_device, Y_size));

    #pragma omp parallel for
    for (int r = 0; r < m; r++) {
        for (int c = 0; c < k; c++) {
            W_host[r * k + c] = W_gen({r, c});
            // W_host[r * k + c] = 1.;
        }
    }

    #pragma omp parallel for
    for (int r = 0; r < k; r++) {
        for (int c = 0; c < n; c++) {
            X_host[r * n + c] = X_gen({r, c});
            // X_host[r * n + c] = 1.;
        }
    }
    checkCudaErrors(hipMemcpy(W_device, W_host, W_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(X_device, X_host, X_size, hipMemcpyHostToDevice));


    GRID = dim3((n + BLOCK.x - 1)/BLOCK.x, (m + BLOCK.y - 1)/BLOCK.y);
    shared_mem = BLOCK.x * BLOCK.y * sizeof(float) * 2;
}

test_matmul::~test_matmul() {
    free(W_host);
    free(X_host);
    free(Y_ground_truth_host);
    free(Y_predict_host);
    checkCudaErrors(hipFree(W_device));
    checkCudaErrors(hipFree(X_device));
    checkCudaErrors(hipFree(Y_ground_truth_device));
    checkCudaErrors(hipFree(Y_predict_device));

    hipblasDestroy(handle);
}


INSTANTIATE_TEST_SUITE_P(
    design,
    test_matmul,
    testing::Values(
        std::make_tuple(
            // 256,
            // 256,
            // 256,
            // 512,
            // 512,
            // 512,
            1024,
            1024,
            1024,
            get_rand_data_gen<float, std::uniform_real_distribution>(-1.f, 1.f),
            get_rand_data_gen<float, std::uniform_real_distribution>(-2.f, 2.f),
            dim3(16, 16)
        )
    )
);


INSTANTIATE_TEST_SUITE_P(
    exhaustive_combine_lite,
    test_matmul,
    testing::Combine(
        testing::Values(1, 2, 3, 8, 64),
        testing::Values(1, 2, 128),
        testing::Values(1, 2, 256),
        testing::Values(
            get_rand_data_gen<float, std::uniform_real_distribution>(-1.f, 1.f)
        ),
        testing::Values(
            get_rand_data_gen<float, std::uniform_real_distribution>(-1.f, 1.f)
        ),
        testing::Values(
            dim3(2, 2),
            dim3(8, 8),
            dim3(16, 16)
            // dim3(32, 32) todo: hipErrorLaunchOutOfResources
        )
    )
);


INSTANTIATE_TEST_SUITE_P(
    exhaustive_combine,
    test_matmul,
    testing::Combine(
        testing::Values(1, 2, 3, 8, 64, 512, 4 * 1024),
        testing::Values(1, 2, 128, 2 * 1024),
        testing::Values(1, 2, 256, 3 * 1023),
        testing::Values(
            get_rand_data_gen<float, std::uniform_real_distribution>(1.f, 2.f)
        ),
        testing::Values(
            get_rand_data_gen<float, std::uniform_real_distribution>(2.f, 3.f)
        ),
        testing::Values(
            dim3(2, 2),
            dim3(8, 8),
            dim3(16, 16)
            // dim3(32, 32) todo: hipErrorLaunchOutOfResources
        )
    )
);

TEST_P(test_matmul, positive){
    std::vector<size_t> W_shape = std::vector<size_t>{size_t(m), size_t(k)};
    Tensor show_W(W_shape, hipMemoryTypeDevice, W_device);
    VLOG(8) << "show W \n" << show_W;

    std::vector<size_t> X_shape = std::vector<size_t>{size_t(k), size_t(n)};
    Tensor show_X(X_shape, hipMemoryTypeDevice, X_device);
    VLOG(8) << "show X \n" << show_X;

    hipblasSgemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n,
        m,
        k,
        &alpha,
        X_device,
        n,
        W_device,
        k,
        &beta,
        Y_ground_truth_device,
        n
    );
    hipMemcpy(Y_ground_truth_host, Y_ground_truth_device, Y_size, hipMemcpyDeviceToHost);

    Tensor gt({size_t(m), size_t(n)}, hipMemoryTypeHost, Y_ground_truth_host);
    VLOG(8) << "show gt \n" << gt;

    kmatmulNaive<<<GRID, BLOCK, shared_mem, hipStreamDefault>>>(
        W_device,
        X_device,
        Y_predict_device,
        m,
        k,
        n
    );
    
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipStreamSynchronize(hipStreamDefault));
    hipMemcpy(Y_predict_host, Y_predict_device, Y_size, hipMemcpyDeviceToHost);

    Tensor pd({size_t(m), size_t(n)}, hipMemoryTypeHost, Y_predict_host);
    VLOG(8) << "show pd \n" << pd;

    for (int r = 0; r < m; r++) {
        for (int c = 0; c < n; c++) {
            ASSERT_LE(
                // abs(Y_predict_host[r * n + c] - Y_ground_truth_host[r * n + c]) / Y_ground_truth_host[r * n + c],
                // 0.01
                abs(Y_predict_host[r * n + c] - Y_ground_truth_host[r * n + c]),
                1e-3
            ) << "\nm: " + std::to_string(m) +\
                 "\nn: " + std::to_string(n) +\
                 "\nk: " + std::to_string(k) +\
                 "\nGRID: " << GRID\
                 << "\nBLOCK: " << BLOCK\
                 << "\nat [" << std::to_string(r) << ", " << std::to_string(c) << "]"
                 << Y_predict_host[r * n + c] << " vs " << Y_ground_truth_host[r * n + c];
        }
    }
}